#include "hip/hip_runtime.h"
/**
 * Requires a CUDA compiler
 */

#include <hip/hip_fp16.h>

#include "../matmul.h"
#include "ozaki.h"

#include <array>
#include <vector>

void test_ozaki_split_correctness(LCG* rng, const double epsilon, const size_t max_splits)
{
    const std::array<size_t, 7> rows_sizes = { 2, 4, 8, 10, 256, 1024, 2048 };
    const std::array<size_t, 5> cols_sizes = { 5, 16, 256, 1024, 4096 };
    for (const size_t rows: rows_sizes)
    {
        for (const size_t cols: cols_sizes)
        {
            const size_t size = rows * cols;
            std::cout << "Testing " << rows << "x" << cols << "=" << size << " split.\n";
            double* matrix = new double[size];
            double* backup = new double[size];
            gen_urand<double>(rng, matrix, size);
            memcpy(backup, matrix, size * sizeof(double));

            // Test split to float
            const auto split_float_matrices = ozaki_split_to_float(rows, cols, matrix, max_splits);
            memset(matrix, 0, size * sizeof(double));
            for (const auto& m: split_float_matrices)
                for (size_t ij = 0; ij < size; ++ij)
                    matrix[ij] += m[ij];

            // Calculate error
            double max_err = 0.0;
            for (size_t ij = 0; ij < size; ++ij)
            {
                const double abs_err = fabs(matrix[ij] - backup[ij]);
                if (abs_err > epsilon)
                {
                    std::cout
                        << "\033[31m" << "[FAILURE] \033[0m (split to float)\n"
                        << "\tOccured at row " << (ij / cols) << ", col " << (ij % cols) << "\n"
                        << "\tExpected: " << backup[ij] << " Actual: " << matrix[ij] << "\n"
                        << "\tAbsolute error: \033[33m" << abs_err << "\033[0m\n";
                    return;
                }
                max_err = fmax(max_err, abs_err);
            }
            std::cout << "Max err (float): " << max_err << "\n";

            // Test split to half
            memcpy(matrix, backup, size * sizeof(double));
            const auto split_half_matrices = ozaki_split_to_half(rows, cols, matrix, max_splits);
            memset(matrix, 0, size * sizeof(double));
            for (const auto& m: split_half_matrices)
                for (size_t ij = 0; ij < size; ++ij)
                    matrix[ij] += __half2float(m[ij]);

            // Calculate error
            max_err = 0.0;
            for (size_t ij = 0; ij < size; ++ij)
            {
                const double abs_err = fabs(matrix[ij] - backup[ij]);
                if (abs_err > epsilon)
                {
                    std::cout
                        << "\033[31m" << "[FAILURE] \033[0m (split to half)\n"
                        << "\tOccured at row " << (ij / cols) << ", col " << (ij % cols) << "\n"
                        << "\tExpected: " << backup[ij] << " Actual: " << matrix[ij] << "\n"
                        << "\tAbsolute error: \033[33m" << abs_err << "\033[0m\n";
                    return;
                }
                max_err = fmax(max_err, abs_err);
            }
            std::cout << "Max err (half): " << max_err << "\n";

            delete[] matrix;
            delete[] backup;
        }
    }

    std::cout
        << "\033[32m" << "[SUCCESS]" << "\033[0m Ozaki splits\n"; // Green text
}
