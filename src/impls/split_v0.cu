#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>

#include "../split.h"

void split_v0(const double *A, void *A16, void *dA16, int M, int N)
{
    half *_A16 = (half *) A16;
    half *_dA16 = (half *) dA16;
    for (int i = 0; i < M * N; i++) {
        _A16[i] = __double2half(A[i]);
        double reconstructed = (double) __half2float(_A16[i]);
        _dA16[i] = __double2half(A[i] - reconstructed);
        
        // printf("A[%d] = %f\n", i, A[i]);
        // printf("A16[%d] = %f\n", i, __half2float(_A16[i]));
        // printf("dA16[%d] = %f\n", i, __half2float(_dA16[i]));
        // printf("A16[%d] + dA16[%d] - A[%d] = %f\n", i, i, i, ((double) __half2float(_A16[i]) + (double) __half2float(_dA16[i])) - A[i] );
    }    
}

void splitf_v0(const float *A, void *A16, void *dA16, int M, int N)
{
    half *_A16 = (half *) A16;
    half *_dA16 = (half *) dA16;
    for (int i = 0; i < M * N; i++) {
        _A16[i] = __float2half(A[i]);
        float reconstructed = __half2float(_A16[i]);
        _dA16[i] = __float2half(A[i] - reconstructed);
    }    
}

void split_Ootomo_v0(const double *A, void *A16, void *dA16, int M, int N)
{
    half *_A16 = (half *) A16;
    half *_dA16 = (half *) dA16;
    for (int i = 0; i < M * N; i++) {
        _A16[i] = __double2half(A[i]);
        double reconstructed = (double) __half2float(_A16[i]);
        _dA16[i] = __double2half((A[i] - reconstructed) * 2048.0);
    }
}

void splitf_Ootomo_v0(const float *A, void *A16, void *dA16, int M, int N)
{
    half *_A16 = (half *) A16;
    half *_dA16 = (half *) dA16;
    for (int i = 0; i < M * N; i++) {
        _A16[i] = __float2half(A[i]);
        float reconstructed = __half2float(_A16[i]);
        _dA16[i] = __float2half((A[i] - reconstructed) * 2048.0);
    }
}

