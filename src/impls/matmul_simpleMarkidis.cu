#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#include "../cuda_utils.h"
#include "../matmul.h"
#include "../profiler.h"
#include "./split_merge_cuda.h"

#include "../timer.h"

#include "matmul_cuda.h"

template<int version, int streamCount, bool useScale>
flop_counts matmul_simpleMarkidis(float *A, float *B, float *C, size_t M, size_t K, size_t N) 
{
    assert((M % 16) == 0);
    assert((K % 16) == 0);
    assert((N % 16) == 0);

    PROFILE_FUNCTION_SEGMENT_START("allocate cpu");

    hipStream_t streams[streamCount];
    for(int i = 0; i < streamCount; i++)
        PRINT_ON_ERROR(hipStreamCreate(&streams[i]));

    constexpr float scale = useScale ? (float) (1 << 11) : 1.0f;
    size_t ASize = M * K * sizeof(half);
    size_t BSize = K * N * sizeof(half);
    size_t CSize = M * N * sizeof(float);
    
    PROFILE_SEGMENTS_SWITCH("allocate gpu");

    half *deviceA[2], *deviceB[2];
    float *deviceC[4];
    float *deviceCMerged;
    float *deviceAFull, *deviceBFull;
    for(int i = 0; i < 2; i++)
    {
        PRINT_ON_ERROR(hipMalloc(&deviceA[i], ASize));
        PRINT_ON_ERROR(hipMalloc(&deviceB[i], BSize));
    }
    for(int i = 0; i < 4; i++)
        PRINT_ON_ERROR(hipMalloc(&deviceC[i], CSize));
    PRINT_ON_ERROR(hipMalloc(&deviceCMerged, CSize));
    PRINT_ON_ERROR(hipMalloc(&deviceAFull, M*K*sizeof(float)));
    PRINT_ON_ERROR(hipMalloc(&deviceBFull, K*N*sizeof(float)));

    PROFILE_SEGMENTS_SWITCH("memcpy h2d & split");

    size_t copyCountA = (M*K)/streamCount;
    size_t copySizeA = copyCountA * sizeof(float);
    size_t copyCountB = (K*N)/streamCount;
    size_t copySizeB = copyCountB * sizeof(float);
    for(int i = 0; i < streamCount; i++)
    {
        size_t offsetA = copyCountA * i;
        size_t offsetB = copyCountB * i;
        PRINT_ON_ERROR(
                hipMemcpyAsync(deviceAFull + offsetA, 
                                 A + offsetA, copySizeA, 
                                 hipMemcpyHostToDevice, streams[i])
        );
        PRINT_ON_ERROR(
                hipMemcpyAsync(deviceBFull + offsetB, 
                                 B + offsetB, copySizeB, 
                                hipMemcpyHostToDevice, streams[i])
        );
        split_2<float, half>
               <<<DivRoundUp(copyCountA, 256), 256, 0, streams[i]>>>
               (deviceAFull + offsetA, deviceA[0] + offsetA, deviceA[1] + offsetA, scale);
        split_2<float, half>
               <<<DivRoundUp(copyCountB, 256), 256, 0, streams[i]>>>
               (deviceBFull + offsetB, deviceB[0] + offsetB, deviceB[1] + offsetB, scale);
    }

    PRINT_ON_ERROR(hipGetLastError());
    CUDA_DEVICE_SYNCHRONIZE();

    PROFILE_SEGMENTS_SWITCH("matmul");
    for(int i = 0; i < 4; i++)
    {
        matmulTensorCores<half, float, version>(deviceA[i/2], deviceB[i%2], deviceC[i], M, K, N);
    }
    CUDA_DEVICE_SYNCHRONIZE();

    PROFILE_SEGMENTS_SWITCH("merge");
    merge_2<float, float, true><<<DivRoundUp(M*N, 256), 256>>>
              (deviceCMerged, deviceC[0], deviceC[1], deviceC[2], deviceC[3], scale);
    PRINT_ON_ERROR(hipGetLastError());
    CUDA_DEVICE_SYNCHRONIZE();

    PROFILE_SEGMENTS_SWITCH("memcpy device2host");
    PRINT_ON_ERROR(hipMemcpy(C, deviceCMerged, CSize, hipMemcpyDeviceToHost));

    PROFILE_SEGMENTS_SWITCH("free");

    for(int i = 0; i < 2; i++)
    {
        PRINT_ON_ERROR(hipFree(deviceA[i]));
        PRINT_ON_ERROR(hipFree(deviceB[i]));
    }
    for(int i = 0; i < 4; i++)
        PRINT_ON_ERROR(hipFree(deviceC[i]));
    PRINT_ON_ERROR(hipFree(deviceCMerged));
    PRINT_ON_ERROR(hipFree(deviceAFull));
    PRINT_ON_ERROR(hipFree(deviceBFull));

    for(int i = 0; i < streamCount; i++)
        PRINT_ON_ERROR(hipStreamDestroy(streams[i]));

    PROFILE_SEGMENT_FUNCTION_END();
/**
 * Flop counts of markidis should be very similar to Ootomo, with the difference that we
 * only require one flop32 for splitting an element and similarly for merging.
 * Furthermore, we perform 4 fp16 matmuls instead of 3
 * 
 * flops16:
 * 4*(2*M*K*N) (4 matmuls)
 * 
 * flops32:
 * M*K + K*N (splitting A and B)
 * + 3*N*M (merging into C)
 */
    flop_counts counts = {8L*M*K*N, M*K + K*N + 3L*N*M, 0L};
    return counts;
}

template<typename Type>
static __global__ 
void divide_cuda(Type *C, int N, double scale)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < N)
        C[i] /= scale;
}

template flop_counts matmul_simpleMarkidis<0, 1, false>(float *A, float *B, float *C, size_t M, size_t K, size_t N);
template flop_counts matmul_simpleMarkidis<1, 1, false>(float *A, float *B, float *C, size_t M, size_t K, size_t N);
template flop_counts matmul_simpleMarkidis<2, 1, false>(float *A, float *B, float *C, size_t M, size_t K, size_t N);
template flop_counts matmul_simpleMarkidis<3, 1, false>(float *A, float *B, float *C, size_t M, size_t K, size_t N);
template flop_counts matmul_simpleMarkidis<3, 4, false>(float *A, float *B, float *C, size_t M, size_t K, size_t N);
template flop_counts matmul_simpleMarkidis<4, 1, false>(float *A, float *B, float *C, size_t M, size_t K, size_t N);
template flop_counts matmul_simpleMarkidis<4, 1, true>(float *A, float *B, float *C, size_t M, size_t K, size_t N);
template flop_counts matmul_simpleMarkidis<5, 1, false>(float *A, float *B, float *C, size_t M, size_t K, size_t N);
template flop_counts matmul_simpleMarkidis<5, 1, true>(float *A, float *B, float *C, size_t M, size_t K, size_t N);

template<int splitCount, int mergeCount, typename mulInputType, typename mulType, typename mulOutputType, bool useTensorCores>
flop_counts matmul_simpleMarkidis_double(double *A, double *B, double *C, size_t M, size_t K, size_t N,
                                        std::pair<int, int> mergePattern[mergeCount], double scale) 
{
    assert((M % 16) == 0);
    assert((K % 16) == 0);
    assert((N % 16) == 0);

    PROFILE_FUNCTION_SEGMENT_START("allocate cpu");

    size_t ASizeH = M * K * sizeof(mulInputType);
    size_t ASizeD = M * K * sizeof(double);
    size_t BSizeH = K * N * sizeof(mulInputType);
    size_t BSizeD = K * N * sizeof(double);
    size_t CSizeF = M * N * sizeof(mulOutputType);
    size_t CSizeD = M * N * sizeof(double);
    
    PROFILE_SEGMENTS_SWITCH("allocate gpu");

    mulInputType *deviceA, *deviceB;
    mulOutputType *deviceC;
    double *deviceCMerged;
    double *deviceAFull, *deviceBFull;
    hipGetLastError();
    PRINT_ON_ERROR(hipMalloc(&deviceA, ASizeH * splitCount));
    PRINT_ON_ERROR(hipMalloc(&deviceB, BSizeH * splitCount));
    PRINT_ON_ERROR(hipMalloc(&deviceC, CSizeF * mergeCount));
    PRINT_ON_ERROR(hipMalloc(&deviceCMerged, CSizeD));
    PRINT_ON_ERROR(hipMalloc(&deviceAFull, ASizeD));
    PRINT_ON_ERROR(hipMalloc(&deviceBFull, BSizeD));

    PROFILE_SEGMENTS_SWITCH("memcpy host2device");

    PRINT_ON_ERROR(hipMemcpy(deviceAFull, A, ASizeD, hipMemcpyHostToDevice));
    PRINT_ON_ERROR(hipMemcpy(deviceBFull, B, BSizeD, hipMemcpyHostToDevice));

    PROFILE_SEGMENTS_SWITCH("split");

    split_cuda_double<splitCount, mulInputType><<<DivRoundUp(M*K, 256), 256>>>(deviceAFull, deviceA, M * K, scale);
    PRINT_ON_ERROR(hipGetLastError());
    split_cuda_double<splitCount, mulInputType><<<DivRoundUp(K*N, 256), 256>>>(deviceBFull, deviceB, K * N, scale);
    PRINT_ON_ERROR(hipGetLastError());

    CUDA_DEVICE_SYNCHRONIZE();

    PROFILE_SEGMENTS_SWITCH("matmul");
    for(int i = 0; i < mergeCount; i++)
    {
        size_t aIndex = mergePattern[i].first * M * K;
        size_t bIndex = mergePattern[i].second * K * N;
        size_t cIndex = i * M * N;
        if constexpr(useTensorCores)
            matmulTensorCores<mulInputType, mulOutputType, 2>(deviceA + aIndex, deviceB + bIndex, deviceC + cIndex, M, K, N);
        else 
            matmulCUDACores<mulInputType, mulType, mulOutputType, 1>(deviceA + aIndex, deviceB + bIndex, deviceC + cIndex, M, K, N);
        
        double factor = std::pow(scale, mergePattern[i].first) * std::pow(scale, mergePattern[i].second);
        if (factor > 1.0)
            divide_cuda<mulOutputType><<<DivRoundUp(M*N, 256), 256>>>(deviceC + cIndex, M*N, factor);
    }
    CUDA_DEVICE_SYNCHRONIZE();

    PROFILE_SEGMENTS_SWITCH("merge");
    merge_cuda_double<mergeCount, mulOutputType><<<DivRoundUp(M*N, 256), 256>>>(deviceC, deviceCMerged, M*N);
    PRINT_ON_ERROR(hipGetLastError());
    CUDA_DEVICE_SYNCHRONIZE();

    PROFILE_SEGMENTS_SWITCH("memcpy device2host");
    PRINT_ON_ERROR(hipMemcpy(C, deviceCMerged, CSizeD, hipMemcpyDeviceToHost));

    PROFILE_SEGMENTS_SWITCH("free");

    PRINT_ON_ERROR(hipFree(deviceA));
    PRINT_ON_ERROR(hipFree(deviceB));
    PRINT_ON_ERROR(hipFree(deviceC));
    PRINT_ON_ERROR(hipFree(deviceCMerged));
    PRINT_ON_ERROR(hipFree(deviceAFull));
    PRINT_ON_ERROR(hipFree(deviceBFull));

    PROFILE_SEGMENT_FUNCTION_END();
/**
 * Flop counts of markidis should be very similar to Ootomo, with the difference that we
 * only require one flop32 for splitting an element and similarly for merging.
 * Furthermore, we perform 4 fp16 matmuls instead of 3
 * 
 * flops16:
 * 4*(2*M*K*N) (4 matmuls)
 * 
 * flops32:
 * M*K + K*N (splitting A and B)
 * + 3*N*M (merging into C)
 */
    flop_counts counts = {8L*M*K*N, M*K + K*N + 3L*N*M, 0L};
    return counts;
}

template<>
flop_counts matmul_simpleMarkidis_double<0>(double *A, double *B, double *C, size_t M, size_t K, size_t N)
{
    std::pair<int, int> merges[] = {{2, 2}, {2, 1}, {1, 2}, {0, 2}, {1, 1}, {2, 0}, {0, 1}, {1, 0}, {0, 0}};
    return matmul_simpleMarkidis_double<3, 9, half, float, float, true>(A, B, C, M, K, N, merges, 1.0);
}

template<>
flop_counts matmul_simpleMarkidis_double<1>(double *A, double *B, double *C, size_t M, size_t K, size_t N)
{
    std::pair<int, int> merges[16];
    for(int i = 0; i < 16; i++)
        merges[i] = {i/4, i%4};
    return matmul_simpleMarkidis_double<4, 16, half, float, float, true>(A, B, C, M, K, N, merges, 1.0);
}

template<>
flop_counts matmul_simpleMarkidis_double<2>(double *A, double *B, double *C, size_t M, size_t K, size_t N)
{
    std::pair<int, int> merges[16];
    for(int i = 0; i < 16; i++)
        merges[i] = {i/4, i%4};
    return matmul_simpleMarkidis_double<4, 16, half, float, double, false>(A, B, C, M, K, N, merges, 1.0);
}

template<>
flop_counts matmul_simpleMarkidis_double<3>(double *A, double *B, double *C, size_t M, size_t K, size_t N)
{
    std::pair<int, int> merges[16];
    for(int i = 0; i < 16; i++)
        merges[i] = {i/4, i%4};
    return matmul_simpleMarkidis_double<4, 16, half, float, double, false>(A, B, C, M, K, N, merges, 1 << 11);
}

template<>
flop_counts matmul_simpleMarkidis_double<4>(double *A, double *B, double *C, size_t M, size_t K, size_t N)
{
    std::pair<int, int> merges[25];
    for(int i = 0; i < 25; i++)
        merges[i] = {i/5, i%5};
    return matmul_simpleMarkidis_double<5, 25, half, float, double, false>(A, B, C, M, K, N, merges, 1 << 11);
}

template<>
flop_counts matmul_simpleMarkidis_double<5>(double *A, double *B, double *C, size_t M, size_t K, size_t N)
{
    std::pair<int, int> merges[] = {{0, 0}, {0, 1}, {1, 0}, {1, 1}};
    return matmul_simpleMarkidis_double<2, 4, float, float, float, false>(A, B, C, M, K, N, merges, 1.0);
}

template<>
flop_counts matmul_simpleMarkidis_double<6>(double *A, double *B, double *C, size_t M, size_t K, size_t N)
{
    std::pair<int, int> merges[] = {{0, 0}, {0, 1}, {1, 0}, {1, 1}};
    return matmul_simpleMarkidis_double<2, 4, float, float, double, false>(A, B, C, M, K, N, merges, 1.0);
}

template<>
flop_counts matmul_simpleMarkidis_double<7>(double *A, double *B, double *C, size_t M, size_t K, size_t N)
{
    std::pair<int, int> merges[] = {{0, 0}, {0, 1}, {1, 0}, {1, 1}};
    return matmul_simpleMarkidis_double<2, 4, float, float, double, false>(A, B, C, M, K, N, merges, 1 << 24);
}

template<>
flop_counts matmul_simpleMarkidis_double<8>(double *A, double *B, double *C, size_t M, size_t K, size_t N)
{
    std::pair<int, int> merges[] = {{0, 0}, {0, 1}, {1, 0}, {1, 1}};
    return matmul_simpleMarkidis_double<2, 4, float, double, double, false>(A, B, C, M, K, N, merges, 1.0);
}






#if SM_VERSION >= 800
template<int splitCount, int mergeCount>
flop_counts matmul_simpleMarkidis_double_double(double *A, double *B, double *C, size_t M, size_t K, size_t N,
                                                std::pair<int, int> mergePattern[mergeCount]) 
{
    assert((M % 16) == 0);
    assert((K % 16) == 0);
    assert((N % 16) == 0);

    PROFILE_FUNCTION_SEGMENT_START("allocate cpu");

    size_t ASizeD = M * K * sizeof(double);
    size_t BSizeD = K * N * sizeof(double);
    size_t CSizeD = M * N * sizeof(double);
    
    PROFILE_SEGMENTS_SWITCH("allocate gpu");

    double *deviceA, *deviceB;
    double *deviceC;
    double *deviceCMerged;
    double *deviceAFull, *deviceBFull;
    hipGetLastError();
    PRINT_ON_ERROR(hipMalloc(&deviceA, ASizeD * splitCount));
    PRINT_ON_ERROR(hipMalloc(&deviceB, BSizeD * splitCount));
    PRINT_ON_ERROR(hipMalloc(&deviceC, CSizeD * mergeCount));
    PRINT_ON_ERROR(hipMalloc(&deviceCMerged, CSizeD));
    PRINT_ON_ERROR(hipMalloc(&deviceAFull, ASizeD));
    PRINT_ON_ERROR(hipMalloc(&deviceBFull, BSizeD));

    PROFILE_SEGMENTS_SWITCH("memcpy host2device");

    PRINT_ON_ERROR(hipMemcpy(deviceAFull, A, ASizeD, hipMemcpyHostToDevice));
    PRINT_ON_ERROR(hipMemcpy(deviceBFull, B, BSizeD, hipMemcpyHostToDevice));

    PROFILE_SEGMENTS_SWITCH("split");

    split_cuda_double_double<splitCount><<<DivRoundUp(M*K, 256), 256>>>(deviceAFull, deviceA, M * K);
    PRINT_ON_ERROR(hipGetLastError());
    split_cuda_double_double<splitCount><<<DivRoundUp(K*N, 256), 256>>>(deviceBFull, deviceB, K * N);
    PRINT_ON_ERROR(hipGetLastError());

    CUDA_DEVICE_SYNCHRONIZE();

    PROFILE_SEGMENTS_SWITCH("matmul");
    for(int i = 0; i < mergeCount; i++)
    {
        size_t aIndex = mergePattern[i].first * M * K;
        size_t bIndex = mergePattern[i].second * K * N;
        size_t cIndex = i * M * N;
        matmulTensorCores<double, double, 2>(deviceA + aIndex, deviceB + bIndex, deviceC + cIndex, M, K, N);
        double scale = std::pow(2048, mergePattern[i].first) * std::pow(2048, mergePattern[i].second);
        divide_cuda<double><<<DivRoundUp(M*N, 256), 256>>>(deviceC + cIndex, M*N, scale);
    }
    CUDA_DEVICE_SYNCHRONIZE();

    PROFILE_SEGMENTS_SWITCH("merge");
    merge_cuda_double<mergeCount><<<DivRoundUp(M*N, 256), 256>>>(deviceC, deviceCMerged, M*N);
    PRINT_ON_ERROR(hipGetLastError());
    CUDA_DEVICE_SYNCHRONIZE();

    PROFILE_SEGMENTS_SWITCH("memcpy device2host");
    PRINT_ON_ERROR(hipMemcpy(C, deviceCMerged, CSizeD, hipMemcpyDeviceToHost));

    PROFILE_SEGMENTS_SWITCH("free");

    PRINT_ON_ERROR(hipFree(deviceA));
    PRINT_ON_ERROR(hipFree(deviceB));
    PRINT_ON_ERROR(hipFree(deviceC));
    PRINT_ON_ERROR(hipFree(deviceCMerged));
    PRINT_ON_ERROR(hipFree(deviceAFull));
    PRINT_ON_ERROR(hipFree(deviceBFull));

    PROFILE_SEGMENT_FUNCTION_END();
/**
 * Flop counts of markidis should be very similar to Ootomo, with the difference that we
 * only require one flop32 for splitting an element and similarly for merging.
 * Furthermore, we perform 4 fp16 matmuls instead of 3
 * 
 * flops16:
 * 4*(2*M*K*N) (4 matmuls)
 * 
 * flops32:
 * M*K + K*N (splitting A and B)
 * + 3*N*M (merging into C)
 */
    flop_counts counts = {8L*M*K*N, M*K + K*N + 3L*N*M, 0L};
    return counts;
}

template<>
flop_counts matmul_simpleMarkidis_double_double<0>(double *A, double *B, double *C, size_t M, size_t K, size_t N)
{
    std::pair<int, int> merges[] = {{2, 2}, {2, 1}, {1, 2}, {0, 2}, {1, 1}, {2, 0}, {0, 1}, {1, 0}, {0, 0}};
    return matmul_simpleMarkidis_double_double<3, 9>(A, B, C, M, K, N, merges);
}

template<>
flop_counts matmul_simpleMarkidis_double_double<1>(double *A, double *B, double *C, size_t M, size_t K, size_t N)
{
    //std::pair<int, int> merges[16];
    //for(int i = 0; i < 16; i++)
        //merges[i] = {i/4, i%4};
    std::pair<int, int> merges[] = {{3, 3}, {3, 2}, {2, 3}, {2, 2}, {3, 1}, {1, 3}, {2, 1}, {1, 2}, {3, 0}, {0, 3}, {2, 0}, {0, 2},  {1, 1}, {0, 1}, {1, 0}, {0, 0}};
    return matmul_simpleMarkidis_double_double<4, 16>(A, B, C, M, K, N, merges);
}
#endif
