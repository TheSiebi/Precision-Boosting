#include <assert.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#include "../split.h"
#include "../cuda_utils.h"
#include "../timer.h"
#include "./matmul_cuda.h"

flop_counts matmul_basic_Ootomo_v0(float *A, float *B, float *C, size_t M, size_t K, size_t N)
{
    // Allocate host memory
    half* A16      = (half*) malloc(M * K * sizeof(half));
    half* dA16     = (half*) malloc(M * K * sizeof(half));
    half* B16      = (half*) malloc(K * N * sizeof(half));
    half* dB16     = (half*) malloc(K * N * sizeof(half));
    float* A16B16  = (float*)malloc(M * N * sizeof(float));
    float* dA16B16 = (float*)malloc(M * N * sizeof(float));
    float* A16dB16 = (float*)malloc(M * N * sizeof(float));

    // Split (host)
    splitf_Ootomo_v0(A, A16, dA16, M, K);
    splitf_Ootomo_v0(B, B16, dB16, K, N);

    // Allocate device memory
    half*  dev_A16;
    half*  dev_dA16;
    half*  dev_B16;
    half*  dev_dB16;
    float* dev_A16B16;
    float* dev_dA16B16;
    float* dev_A16dB16;
    PRINT_ON_ERROR(hipMalloc((void**)&dev_A16,     M * K * sizeof(half)));
    PRINT_ON_ERROR(hipMalloc((void**)&dev_dA16,    M * K * sizeof(half)));
    PRINT_ON_ERROR(hipMalloc((void**)&dev_B16,     K * N * sizeof(half)));
    PRINT_ON_ERROR(hipMalloc((void**)&dev_dB16,    K * N * sizeof(half)));
    PRINT_ON_ERROR(hipMalloc((void**)&dev_A16B16,  M * N * sizeof(float)));
    PRINT_ON_ERROR(hipMalloc((void**)&dev_dA16B16, M * N * sizeof(float)));
    PRINT_ON_ERROR(hipMalloc((void**)&dev_A16dB16, M * N * sizeof(float)));

    // Copy from host to device
    PRINT_ON_ERROR(hipMemcpy(dev_A16,     A16,     M * K * sizeof(half), hipMemcpyHostToDevice));
    PRINT_ON_ERROR(hipMemcpy(dev_dA16,    dA16,    M * K * sizeof(half), hipMemcpyHostToDevice));
    PRINT_ON_ERROR(hipMemcpy(dev_B16,     B16,     K * N * sizeof(half), hipMemcpyHostToDevice));
    PRINT_ON_ERROR(hipMemcpy(dev_dB16,    dB16,    K * N * sizeof(half), hipMemcpyHostToDevice));

    // Multiply matrices
    matmulCUDACores<half, float, 0>(dev_A16, dev_B16, dev_A16B16, M, K, N);
    PRINT_ON_ERROR(hipGetLastError());
    matmulCUDACores<half, float, 0>(dev_dA16, dev_B16, dev_dA16B16, M, K, N);
    PRINT_ON_ERROR(hipGetLastError());
    matmulCUDACores<half, float, 0>(dev_A16, dev_dB16, dev_A16dB16, M, K, N);
    PRINT_ON_ERROR(hipGetLastError());

    // Copy from device to host
    PRINT_ON_ERROR(hipDeviceSynchronize());
    PRINT_ON_ERROR(hipMemcpy(A16B16, dev_A16B16, M * N * sizeof(float), hipMemcpyDeviceToHost));
    PRINT_ON_ERROR(hipMemcpy(dA16B16, dev_dA16B16, M * N * sizeof(float), hipMemcpyDeviceToHost));
    PRINT_ON_ERROR(hipMemcpy(A16dB16, dev_A16dB16, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // Accumulate (host)
    for (int i = 0; i < M * N; ++i)
    {
        const float ab = A16B16[i];
        const float dab = dA16B16[i];
        const float adb = A16dB16[i];
        C[i] = ab + (dab + adb) / 2048.f;
    }

    // Free device memory
    PRINT_ON_ERROR(hipFree(dev_A16));
    PRINT_ON_ERROR(hipFree(dev_dA16));
    PRINT_ON_ERROR(hipFree(dev_B16));
    PRINT_ON_ERROR(hipFree(dev_dB16));
    PRINT_ON_ERROR(hipFree(dev_A16B16));
    PRINT_ON_ERROR(hipFree(dev_dA16B16));
    PRINT_ON_ERROR(hipFree(dev_A16dB16));

    // Free host memory
    free(A16);
    free(dA16);
    free(B16);
    free(dB16);
    free(A16B16);
    free(dA16B16);
    free(A16dB16);

    flop_counts counts = {3L*M*K*N, 2L*M*K + 2L*K*N + 3L*M*K*N + 3L*M*N, 0L};
    return counts;
}
