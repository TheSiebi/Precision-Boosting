#include "hip/hip_runtime.h"
#include "ozaki.h"
#include <cstdint>
#include <cstdio>

#include <algorithm>
#include <stdexcept>
#include <string>
#include <vector>

#include "../timer.h"
#include "../profiler.h"

#include "../matmul.h"

#include <hip/hip_fp16.h>

#define MAX_SPLITS 10

size_t ix(size_t row, size_t col, size_t rows, size_t cols)
{
    return col + row * cols;
}

// Turns a [rows, cols] matrix into a [cols, rows] matrix
template<class T>
void transpose(const size_t rows, const size_t cols, T* data)
{
    T* backup = (T*) malloc(rows * cols * sizeof(T));
    memcpy(backup, data, rows * cols * sizeof(T));

    for (size_t i = 0; i < cols; ++i)
        for (size_t j = 0; j < rows; ++j)
            data[ix(i, j, cols, rows)] = backup[ix(j, i, rows, cols)];

    free(backup);
}

template<class T>
void matmul_triple_loop(const size_t m, const size_t k, const size_t n, const T* a, const T* b, T* c)
{
    for (size_t row = 0; row < m; ++row)
    {
        for (size_t col = 0; col < n; ++col)
        {
            T sum = (T) 0;
            for (size_t l = 0; l < k; ++l)
                sum += a[ix(row, l, m, k)] * b[ix(l, col, k, n)];
            c[ix(row, col, m, n)] = sum;
        }
    }
}

std::vector<std::vector<half>> ozaki_split_to_half(const size_t m, const size_t n, double* a, const int l)
{
    // q = size(A, 2);
    // This simply means q := n
    const int q = n;

    // k = 1;
    // Keeps consistency with paper. We'll subtract one every time we index into D.
    int k = 1;

    // beta = fl(...)
    const double log2u = -11.f; // half precision
    const double beta = ceil((-log2u + log2(q)) / 2.0);

    // D{1} = zeros(size(A));
    std::vector<std::vector<half>> D = { std::vector<half>(m * n, __float2half(0.f)) };

    // while(k < l)
    while (k < l)
    {
        // mu = max(abs(A), [], 2);
        std::vector<double> mu(m, 0.0);
        for (size_t i = 0; i < m; ++i)
            for (size_t j = 0; j < q; ++j)
                mu[i] = fmax(mu[i], fabs(a[ix(i, j, m, q)]));

        // if(max(mu) == 0) -> return
        double max = 0.0;
        for (const auto mu_i: mu)
            max = fmax(max, mu_i);
        if (max == 0.0)
        {
            // printf("Early termination\n");
            return D;
        }

        // w = fl(...);
        std::vector<double> w(m);
        for (size_t i = 0; i < m; ++i)
            w[i] = exp2(ceil(log2(mu[i])) + beta);

        // S = repmat(w, 1, q);
        std::vector<double> S(m * n);
        for (size_t i = 0; i < m; ++i)
            for (size_t j = 0; j < n; ++j)
                S[ix(i, j, m, n)] = w[ix(i, 0, m, 1)];

        // D{k} = fl((A + S) - S);
        // A = fl(A - D{k});
        D.resize(k, std::vector<half>(m * n));
        for (size_t ij = 0; ij < m * n; ++ij)
        {
            // Note: unclear from paper whether ((A+S)-S) computation should happen as double or half
            double intermediate = a[ij] + S[ij];
            asm volatile("" : : "r,m"(intermediate) : "memory"); // avoid compiler optimizations
            intermediate -= S[ij];
            D[k - 1][ij] = __float2half((float) intermediate);
            a[ij] -= __half2float(D[k - 1][ij]);
        }

        // % Checking sparsity of D{k}
        // Omitted

        // k = k + 1;
        ++k;
    }

    // if(k == l)
    if (k == l)
    {
        // Happens if early termination criterion was not met.
        // D{k} = A;
        // printf("Early termination not reached for k = %zu = l, D.size() = %zu\n", k, D.size());
        D.resize(k, std::vector<half>(m * n));
        for (size_t ij = 0; ij < m * n; ++ij)
            D[k - 1][ij] = __float2half((float) a[ij]); // Downcasting? Paper just says D{k} = A
    }

    return D;
}

/**
 * Implementation of Algorithm 3 in Ozaki paper.
 * Returns an unevaluated sum as a vector of matrices stored as vectors.
 * Uses fp32 (float) to emulate fp64 (double) precision.
 * Completely disregards sparsity criterion.
 */
std::vector<std::vector<float>> ozaki_split_to_float(const size_t m, const size_t n, double* a, const int l)
{
    // q = size(A, 2);
    // This simply means q := n
    const int q = n;

    // k = 1;
    // Keeps consistency with paper. We'll subtract one every time we index into D.
    int k = 1;

    // beta = fl(...)
    const float log2u = -24.f;
    const float beta = ceilf((-log2u + log2f(q)) / 2.f);

    // D{1} = zeros(size(A));
    std::vector<std::vector<float>> D = { std::vector<float>(m * n, 0.f) };

    // while(k < l)
    while (k < l)
    {
        // mu = max(abs(A), [], 2);
        std::vector<double> mu(m, 0.0);
        for (size_t i = 0; i < m; ++i)
            for (size_t j = 0; j < q; ++j)
                mu[i] = fmax(mu[i], fabs(a[ix(i, j, m, q)]));

        // if(max(mu) == 0) -> return
        double max = 0.0;
        for (const auto mu_i: mu)
            max = fmax(max, mu_i);
        if (max == 0.0)
        {
            // printf("Early termination\n");
            return D;
        }

        // w = fl(...);
        std::vector<float> w(m);
        for (size_t i = 0; i < m; ++i)
            w[i] = exp2f(ceilf((float) log2f(mu[i])) + beta);

        // S = repmat(w, 1, q);
        std::vector<float> S(m * n);
        for (size_t i = 0; i < m; ++i)
            for (size_t j = 0; j < n; ++j)
                S[ix(i, j, m, n)] = w[ix(i, 0, m, 1)];

        // D{k} = fl((A + S) - S);
        // A = fl(A - D{k});
        D.resize(k, std::vector<float>(m * n));
        for (size_t ij = 0; ij < m * n; ++ij)
        {
            D[k - 1][ij] = a[ij] + S[ij];
            D[k - 1][ij] -= S[ij];
            a[ij] -= D[k - 1][ij];
        }

        // % Checking sparsity of D{k}
        // Omitted

        // k = k + 1;
        ++k;
    }

    // if(k == l)
    if (k == l)
    {
        // Happens if early termination criterion was not met.
        // D{k} = A;
        // printf("Early termination not reached for k = %zu = l, D.size() = %zu\n", k, D.size());
        D.resize(k, std::vector<float>(m * n));
        for (size_t ij = 0; ij < m * n; ++ij)
            D[k - 1][ij] = (float) a[ij]; // Downcasting? Paper just says D{k} = A
    }

    return D;

}


/**
 * Implementation of Algorithm 4 in Ozaki paper.
 * Returns an unevaluated sum as a vector of matrices stored as vectors.
 * Completely disregards sparsity criterion.
 */
template<int version>
std::vector<std::vector<float>> ozaki_mul(const size_t m, const size_t n, const size_t p, double* a, double* b, int64_t* nA_ptr, int64_t* nB_ptr)
{
    // [m, n] = size(A); [n, p] = size(B);
    // Given as parameters

    if constexpr (version == 0 || version == 1)
    {
        // D = Split_Mat(A, inf, delta); nA = length(D);
        auto D = ozaki_split_to_float(m, n, a, MAX_SPLITS);
        const auto nA = D.size();
        *nA_ptr = (int) nA;

        // E = Split_Mat(BT, inf, delta); nB = length(E);
        // Do we really need to transpose B?
        // transpose(n, p, b);
        auto E = ozaki_split_to_float(n, p, b, MAX_SPLITS);
        const auto nB = E.size();
        *nB_ptr = (int) nB;

        // for r = 1 : nB, E{r} = E{r}T ; end
        // again, why transpose?
        // for (auto& matrix: E)
        //     transpose(p, n, matrix.data());

        int t = 0;
        std::vector<std::vector<float>> C(nA * nB, std::vector<float>(m * p));
        for (int r = 0; r < nA; ++r)
        {
            for (int s = 0; s < nB; ++s)
            {
                if constexpr (version == 0)
                    matmul_triple_loop<float>(m, n, p, D[r].data(), E[s].data(), C[t++].data());
                else if constexpr (version == 1)
                    matmul_cuda<float, float, 1, false>(D[r].data(), E[s].data(), C[t++].data(), m, n, p);
                else
                    throw std::runtime_error(std::string("unimplemented ozaki version " + version));
            }
        }

        return C;
    }

    else if constexpr (version == 2)
    {
        // D = Split_Mat(A, inf, delta); nA = length(D);
        auto D = ozaki_split_to_half(m, n, a, MAX_SPLITS);
        const auto nA = D.size();
        *nA_ptr = (int) nA;

        // E = Split_Mat(BT, inf, delta); nB = length(E);
        auto E = ozaki_split_to_half(n, p, b, MAX_SPLITS);
        const auto nB = E.size();
        *nB_ptr = (int) nB;

        int t = 0;
        std::vector<std::vector<float>> C(nA * nB, std::vector<float>(m * p));
        for (int r = 0; r < nA; ++r)
            for (int s = 0; s < nB; ++s)
                matmul_cuda<half, float, 3, true>(D[r].data(), E[s].data(), C[t++].data(), m, n, p);

        return C;
    }

    else
        throw std::runtime_error(std::string("unimplemented ozaki version " + version));

}

// Ozaki paper uses A [m, n] and B [n, p] matrices
template<int version>
flop_counts matmul_ozaki(double *a, double *b, double *c, size_t m, size_t n, size_t p)
{
    // Ozaki splitting modifies input matrices. Therefore, copies must be made.
    std::vector<double> a_copy(a, a + m * n);
    std::vector<double> b_copy(b, b + n * p);

    // Splitting configuration (nA, nB) influences flop-count, and must be retrieved.
    int64_t nA, nB;

    PROFILE_FUNCTION_START();
    const auto unevaluated_sum = ozaki_mul<version>(m, n, p, a_copy.data(), b_copy.data(), &nA, &nB);
    // std::cout << "nA: " << nA << "/" << MAX_SPLITS << ", ";
    // std::cout << "nB: " << nB << "/" << MAX_SPLITS << "\n";
    memset(c, 0, m * p * sizeof(double));
    for (size_t ij = 0; ij < m * p; ++ij)
        for (const auto& matrix: unevaluated_sum)
            c[ij] += matrix[ij];

    PROFILE_FUNCTION_END();

    flop_counts counts =
    {
        0L,
        8L + (4L*m+3L*m*n)*nA + (4L*n+3L*n*p)*nB + 2L*nA*nB*m*n*p,
        (2L*m*n+m)*nA + (2L*n*p+n)*nB + m*p*nA*nB
    };
    return counts;
}

template flop_counts matmul_ozaki<0>(double *a, double *b, double *c, size_t m, size_t n, size_t p);
template flop_counts matmul_ozaki<1>(double *a, double *b, double *c, size_t m, size_t n, size_t p);
template flop_counts matmul_ozaki<2>(double *a, double *b, double *c, size_t m, size_t n, size_t p);
