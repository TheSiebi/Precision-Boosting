#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#include "../split.h"
#include "../cuda_utils.h"
#include "../timer.h"

__global__ void basic_mixed_precision_matmul(const half* A, const half* B, float *C, int M, int K, int N)
{
    const auto index = [](int row, int col, int rows, int cols)
    {
        return row * cols + col; // row-major
    };
    const int tid = blockIdx.x;
    const int row = tid / N;
    const int col = tid % N;
    C[index(row, col, M, N)] = 0.f;
    for (int l = 0; l < K; ++l)
        // C[index(row, col, M, N)] += (float)(A[index(row, l, M, K)] * B[index(l, col, K, N)]);
        C[index(row, col, M, N)] += __half2float(A[index(row, l, M, K)] * B[index(l, col, K, N)]);
}

flop_counts matmul_simpleOotomo_v0(float *A, float *B, float *C, int M, int K, int N)
{
    // Allocate host memory
    half* A16      = (half*) malloc(M * K * sizeof(half));
    half* dA16     = (half*) malloc(M * K * sizeof(half));
    half* B16      = (half*) malloc(K * N * sizeof(half));
    half* dB16     = (half*) malloc(K * N * sizeof(half));
    float* A16B16  = (float*)malloc(M * N * sizeof(float));
    float* dA16B16 = (float*)malloc(M * N * sizeof(float));
    float* A16dB16 = (float*)malloc(M * N * sizeof(float));

    // Split (host)
    splitf_Ootomo_v0(A, A16, dA16, M, K);
    splitf_Ootomo_v0(B, B16, dB16, K, N);

    // Allocate device memory
    half*  dev_A16;
    half*  dev_dA16;
    half*  dev_B16;
    half*  dev_dB16;
    float* dev_A16B16;
    float* dev_dA16B16;
    float* dev_A16dB16;
    PRINT_ON_ERROR(hipMalloc((void**)&dev_A16,     M * K * sizeof(half)));
    PRINT_ON_ERROR(hipMalloc((void**)&dev_dA16,    M * K * sizeof(half)));
    PRINT_ON_ERROR(hipMalloc((void**)&dev_B16,     K * N * sizeof(half)));
    PRINT_ON_ERROR(hipMalloc((void**)&dev_dB16,    K * N * sizeof(half)));
    PRINT_ON_ERROR(hipMalloc((void**)&dev_A16B16,  M * N * sizeof(float)));
    PRINT_ON_ERROR(hipMalloc((void**)&dev_dA16B16, M * N * sizeof(float)));
    PRINT_ON_ERROR(hipMalloc((void**)&dev_A16dB16, M * N * sizeof(float)));

    // Copy from host to device
    PRINT_ON_ERROR(hipMemcpy(dev_A16,     A16,     M * K * sizeof(half), hipMemcpyHostToDevice));
    PRINT_ON_ERROR(hipMemcpy(dev_dA16,    dA16,    M * K * sizeof(half), hipMemcpyHostToDevice));
    PRINT_ON_ERROR(hipMemcpy(dev_B16,     B16,     K * N * sizeof(half), hipMemcpyHostToDevice));
    PRINT_ON_ERROR(hipMemcpy(dev_dB16,    dB16,    K * N * sizeof(half), hipMemcpyHostToDevice));

    // Multiply matrices
    basic_mixed_precision_matmul<<<M * N, 1>>>(dev_A16, dev_B16, dev_A16B16, M, K, N);
    PRINT_ON_ERROR(hipGetLastError());
    basic_mixed_precision_matmul<<<M * N, 1>>>(dev_dA16, dev_B16, dev_dA16B16, M, K, N);
    PRINT_ON_ERROR(hipGetLastError());
    basic_mixed_precision_matmul<<<M * N, 1>>>(dev_A16, dev_dB16, dev_A16dB16, M, K, N);
    PRINT_ON_ERROR(hipGetLastError());

    // Copy from device to host
    PRINT_ON_ERROR(hipDeviceSynchronize());
    PRINT_ON_ERROR(hipMemcpy(A16B16, dev_A16B16, M * N * sizeof(float), hipMemcpyDeviceToHost));
    PRINT_ON_ERROR(hipMemcpy(dA16B16, dev_dA16B16, M * N * sizeof(float), hipMemcpyDeviceToHost));
    PRINT_ON_ERROR(hipMemcpy(A16dB16, dev_A16dB16, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // Accumulate (host)
    for (int i = 0; i < M * N; ++i)
    {
        const float ab = A16B16[i];
        const float dab = dA16B16[i];
        const float adb = A16dB16[i];
        C[i] = ab + (dab + adb) / 2048.f;
    }

    // Free device memory
    PRINT_ON_ERROR(hipFree(dev_A16));
    PRINT_ON_ERROR(hipFree(dev_dA16));
    PRINT_ON_ERROR(hipFree(dev_B16));
    PRINT_ON_ERROR(hipFree(dev_dB16));
    PRINT_ON_ERROR(hipFree(dev_A16B16));
    PRINT_ON_ERROR(hipFree(dev_dA16B16));
    PRINT_ON_ERROR(hipFree(dev_A16dB16));

    // Free host memory
    free(A16);
    free(dA16);
    free(B16);
    free(dB16);
    free(A16B16);
    free(dA16B16);
    free(A16dB16);

    flop_counts counts = {3L*M*K*N, 2L*M*K + 2L*K*N + 3L*M*K*N + 3L*M*N, 0L};
    return counts;
}
