#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>

#include "../matmul.h"
#include "../profiler.h"
#include "../cuda_utils.h"
#include "../timer.h"

__global__ void matmul_cuda_v0_kernel(double *A, double *B, double *C, int M, int K, int N) 
{
    int m = blockIdx.x * blockDim.x + threadIdx.x;
    int n = blockIdx.y * blockDim.y + threadIdx.y;
    double result = 0.0;
    for (int k = 0; k < K; k++) 
    {
        result += A[m*K + k] * B[k*N + n];
    }
    C[m*N + n] = result;
}


flop_counts matmul_cuda_v0(double *A, double *B, double *C, int M, int K, int N) 
{
    assert((M & 0xF) == 0);
    assert((K & 0xF) == 0);
    assert((N & 0xF) == 0);

    PROFILE_FUNCTION_SEGMENT_START("allocate");
    size_t ASize = M * K * sizeof(double);
    size_t BSize = K * N * sizeof(double);
    size_t CSize = M * N * sizeof(double);

    double *deviceA, *deviceB, *deviceC;
    PRINT_ON_ERROR(hipMalloc(&deviceA, ASize));
    PRINT_ON_ERROR(hipMalloc(&deviceB, BSize));
    PRINT_ON_ERROR(hipMalloc(&deviceC, CSize));

    PROFILE_SEGMENTS_SWITCH("memcpy host2device");
    PRINT_ON_ERROR(hipMemcpy(deviceA, A, ASize, hipMemcpyHostToDevice));
    PRINT_ON_ERROR(hipMemcpy(deviceB, B, BSize, hipMemcpyHostToDevice));

    PROFILE_SEGMENTS_SWITCH("matmul");
    dim3 threadsPerBlock(16, 16);
    dim3 blocks(M/threadsPerBlock.x, N/threadsPerBlock.y);
    matmul_cuda_v0_kernel<<<blocks, threadsPerBlock>>>(deviceA, deviceB, deviceC, M, K, N);
    PRINT_ON_ERROR(hipGetLastError());

    PRINT_ON_ERROR(hipDeviceSynchronize());

    PROFILE_SEGMENTS_SWITCH("memcpy device2host");
    PRINT_ON_ERROR(hipMemcpy(C, deviceC, CSize, hipMemcpyDeviceToHost));

    PROFILE_SEGMENTS_SWITCH("free");
    PRINT_ON_ERROR(hipFree(deviceA));
    PRINT_ON_ERROR(hipFree(deviceB));
    PRINT_ON_ERROR(hipFree(deviceC));
    PROFILE_SEGMENT_FUNCTION_END();

    flop_counts counts = {0L, 0L, 0L};
    return counts;
}

