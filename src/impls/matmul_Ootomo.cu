#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#include "../matmul.h"
#include "../profiler.h"

/**
 * Note: Kernels in this file have been inspired by: 
 *  - https://github.com/siboehm/SGEMM_CUDA/tree/master
 *  - https://github.com/Bruce-Lee-LY/cuda_hgemm/blob/master/src/wmma/wmma_base.cu#L86
 */


#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

#define WARP_SIZE 32

struct split
{
    // original terms
    half2 x;
    half2 y;
    // error terms
    half2 dx;
    half2 dy;
};

__device__ struct split split_Ootomo(float4 value)
{
    float2 first = make_float2(value.x, value.y);
    float2 second = make_float2(value.z, value.w);
    struct split split;

    split.x = __float22half2_rn(first);
    split.y = __float22half2_rn(second);

    float2 reconstructed = __half22float2(split.x);
    split.dx = __float22half2_rn(make_float2((first.x - reconstructed.x) * 2048, (first.y - reconstructed.y) * 2048));

    reconstructed = __half22float2(split.y);
    split.dy = __float22half2_rn(make_float2((second.x - reconstructed.x) * 2048, (second.y - reconstructed.y) * 2048));

    return split;
}   

__global__ void split_cuda(float *A, half *A0, half *A1)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float value = A[i];
    half mainPart = (half)value;
    A0[i] = mainPart;
    A1[i] = (half)((value - (float)mainPart) * 2048.0f);
}

__global__ void merge_cuda(float *C, float *AB, float *dAB, float *AdB, float *dAdB)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    C[i] = AB[i] + (dAB[i] + AdB[i]) / 2048.0f + dAdB[i] / 4194304.0f;
}


/**
 * Kernel that performs half multiplication using tensore cores. Does not implement
 * specific Ootomo logic. 
 */
template <const int BM, const int BN, const int BK, const int WM, const int WN, const int CHUNK_K,
          const int N_WARP_ROWS_PER_BLOCK,
          const int N_WARP_COLS_PER_BLOCK,
          const int N_WMMA_ROWS_PER_WARP,
          const int N_WMMA_COLS_PER_WARP>
__global__ void matmul_v1_kernel(const half *A, const half *B, float *C, int M, int K, int N)
{
    using namespace nvcuda;

    // allocate space for the current blocktile in shared memory
    __shared__ half As[BM * BK];
    __shared__ half Bs[BK * BN];

    // Move blocktile to beggining of A's row and B's column
    const int cRow = blockIdx.x;
    const int cCol = blockIdx.y;
    A += cRow * BM * K;
    B += cCol * BN;
    C += cRow * BM * N + cCol * BN;

    // warpID in threadBlock
    const int warpID = threadIdx.x / WARP_SIZE;

    // The indices this warp has in the block tile
    const int warpRow = warpID / N_WARP_COLS_PER_BLOCK;
    const int warpCol = warpID % N_WARP_COLS_PER_BLOCK;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> bFrag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> cFrag[N_WMMA_ROWS_PER_WARP][N_WMMA_COLS_PER_WARP];

    for (int i = 0; i < N_WMMA_ROWS_PER_WARP; i++)
    {
        for (int j = 0; j < N_WMMA_COLS_PER_WARP; j++)
        {
            wmma::fill_fragment(cFrag[i][j], 0.0f);
        }
    }

    // Calculate indices that this thread will load from GMEM to SMEM
    // Loads are vectorized and each thread will load 4 elements into SMEM
    // Note that for coalescing, it's important that consecutive threadIDs
    // access consecutive memory addresses
    const int innerRowA = threadIdx.x / (BK / 4);
    const int innerColA = threadIdx.x % (BK / 4);
    const int innerRowB = threadIdx.x / (BN / 4);
    const int innerColB = threadIdx.x % (BN / 4);
    // complete #rows that gets loaded in one loading iteration
    const int strideA = 4 * blockDim.x / BK;
    const int strideB = 4 * blockDim.x / BN;

    // Loop over all block tiles
    for (int bkIdx = 0; bkIdx < K; bkIdx += BK)
    {
        // populate SMEM cache using vectorized loads
        for (int loadOffset = 0; loadOffset < BM; loadOffset += strideA)
        {
            reinterpret_cast<half2 *>(&As[(innerRowA + loadOffset) * BK + innerColA * 4])[0] = 
                reinterpret_cast<const half2 *>(&A[(innerRowA + loadOffset) * K + innerColA * 4])[0];
            reinterpret_cast<half2 *>(&As[(innerRowA + loadOffset) * BK + innerColA * 4 + 2])[0] = 
                reinterpret_cast<const half2 *>(&A[(innerRowA + loadOffset) * K + innerColA * 4 + 2])[0];
        }
        for (int loadOffset = 0; loadOffset < BK; loadOffset += strideB)
        {
            reinterpret_cast<half2 *>(&Bs[(innerRowB + loadOffset) * BN + innerColB * 4])[0] = 
                reinterpret_cast<const half2 *>(&B[(innerRowB + loadOffset) * N + innerColB * 4])[0];
            reinterpret_cast<half2 *>(&Bs[(innerRowB + loadOffset) * BN + innerColB * 4 + 2])[0] = 
                reinterpret_cast<const half2 *>(&B[(innerRowB + loadOffset) * N + innerColB * 4 + 2])[0];
        }

        __syncthreads();

        // advance blocktile
        A += BK;
        B += BK * N;

        // start of data belonging to respective warp
        half *warpAs = &As[warpRow * WM * BK];
        half *warpBs = &Bs[warpCol * WN];
        
        // calculate mmul
        for (int tileRow = 0; tileRow < N_WMMA_ROWS_PER_WARP; tileRow++)
        {
            for (int tileCol = 0; tileCol < N_WMMA_COLS_PER_WARP; tileCol++)
            {
                for (int chunk = 0; chunk < CHUNK_K; chunk++)
                {
                    wmma::load_matrix_sync(aFrag, warpAs + chunk * WMMA_K, BK);
                    wmma::load_matrix_sync(bFrag, warpBs + chunk * WMMA_K * BN, BN);

                    wmma::mma_sync(cFrag[tileRow][tileCol], aFrag, bFrag, cFrag[tileRow][tileCol]);
                }
                warpBs += WMMA_N;
            }
            warpBs = &Bs[warpCol * WN];
            warpAs += WMMA_M * BK;
        }
    }

    // Store results back to C matrix
    float *warpC = &C[warpRow * WM * N + warpCol * WN];
    for (int tileRow = 0; tileRow < N_WMMA_ROWS_PER_WARP; tileRow++)
    {
        for (int tileCol = 0; tileCol < N_WMMA_COLS_PER_WARP; tileCol++)
        {
            wmma::store_matrix_sync(warpC + tileCol * WMMA_N, cFrag[tileRow][tileCol], N, wmma::mem_row_major);
        }
        warpC += WMMA_M * N;
    }
}

// Code adapted from: https://github.com/siboehm/SGEMM_CUDA/tree/master
template <const int BM, const int BN, const int BK, const int WM, const int WN, const int CHUNK_K,
          const int N_WARP_ROWS_PER_BLOCK,
          const int N_WARP_COLS_PER_BLOCK,
          const int N_WMMA_ROWS_PER_WARP,
          const int N_WMMA_COLS_PER_WARP>
__global__ void matmul_v2_kernel(const float *A, const float *B, float *C, int M, int K, int N)
{
    using namespace nvcuda;

    // allocate space for the current blocktile in shared memory
    __shared__ half As[BM * BK];
    __shared__ half Bs[BK * BN];
    __shared__ half dAs[BM * BK];
    __shared__ half dBs[BK * BN];

    // Move blocktile to beggining of A's row and B's column
    const int cRow = blockIdx.x;
    const int cCol = blockIdx.y;
    A += cRow * BM * K;
    B += cCol * BN;
    C += cRow * BM * N + cCol * BN;

    // warpID in threadBlock
    const int warpID = threadIdx.x / WARP_SIZE;
    // thread LaneID in warp
    // const int laneID = threadIdx.x % WARP_SIZE;
    // The indices this warp has in the block tile
    const int warpRow = warpID / N_WARP_COLS_PER_BLOCK;
    const int warpCol = warpID % N_WARP_COLS_PER_BLOCK;

    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> bFrag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> cFrag[N_WMMA_ROWS_PER_WARP][N_WMMA_COLS_PER_WARP];

    for (int i = 0; i < N_WMMA_ROWS_PER_WARP; i++)
    {
        for (int j = 0; j < N_WMMA_COLS_PER_WARP; j++)
        {
            wmma::fill_fragment(cFrag[i][j], 0.0f);
        }
    }

    // Calculate indices that this thread will load from GMEM to SMEM
    // Loads are vectorized and each thread will load 4 elements into SMEM
    // Note that for coalescing, it's important that consecutive threadIDs
    // access consecutive memory addresses
    const int innerRowA = threadIdx.x / (BK / 4);
    const int innerColA = threadIdx.x % (BK / 4);
    const int innerRowB = threadIdx.x / (BN / 4);
    const int innerColB = threadIdx.x % (BN / 4);
    // complete #rows that gets loaded in one loading iteration
    const int strideA = 4 * blockDim.x / BK;
    const int strideB = 4 * blockDim.x / BN;

    // Loop over all block tiles
    for (int bkIdx = 0; bkIdx < K; bkIdx += BK)
    {
        // populate SMEM cache using vectorized loads
        for (int loadOffset = 0; loadOffset < BM; loadOffset += strideA)
        {
            float4 tmp = *(reinterpret_cast<const float4 *>(A + (innerRowA + loadOffset) * K + innerColA * 4));
            struct split tmp_split = split_Ootomo(tmp);
            As[(innerRowA + loadOffset) * BK + innerColA * 4 + 0] = tmp_split.x.x;
            As[(innerRowA + loadOffset) * BK + innerColA * 4 + 1] = tmp_split.x.y;
            As[(innerRowA + loadOffset) * BK + innerColA * 4 + 2] = tmp_split.y.x;
            As[(innerRowA + loadOffset) * BK + innerColA * 4 + 3] = tmp_split.y.y;
            dAs[(innerRowA + loadOffset) * BK + innerColA * 4 + 0] = tmp_split.dx.x;
            dAs[(innerRowA + loadOffset) * BK + innerColA * 4 + 1] = tmp_split.dx.y;
            dAs[(innerRowA + loadOffset) * BK + innerColA * 4 + 2] = tmp_split.dy.x;
            dAs[(innerRowA + loadOffset) * BK + innerColA * 4 + 3] = tmp_split.dy.y;
        }
        for (int loadOffset = 0; loadOffset < BK; loadOffset += strideB)
        {
            float4 tmp = *(reinterpret_cast<const float4 *>(B + (innerRowB + loadOffset) * N + innerColB * 4));
            struct split tmp_split = split_Ootomo(tmp);
            Bs[(innerRowB + loadOffset) * BN + innerColB * 4 + 0] = tmp_split.x.x;
            Bs[(innerRowB + loadOffset) * BN + innerColB * 4 + 1] = tmp_split.x.y;
            Bs[(innerRowB + loadOffset) * BN + innerColB * 4 + 2] = tmp_split.y.x;
            Bs[(innerRowB + loadOffset) * BN + innerColB * 4 + 3] = tmp_split.y.y;
            dBs[(innerRowB + loadOffset) * BN + innerColB * 4 + 0] = tmp_split.dx.x;
            dBs[(innerRowB + loadOffset) * BN + innerColB * 4 + 1] = tmp_split.dx.y;
            dBs[(innerRowB + loadOffset) * BN + innerColB * 4 + 2] = tmp_split.dy.x;
            dBs[(innerRowB + loadOffset) * BN + innerColB * 4 + 3] = tmp_split.dy.y;
        }

        __syncthreads();

        // advance blocktile
        A += BK;
        B += BK * N;

        // start of data belonging to respective warp
        half *warpAs = &As[warpRow * WM * BK];
        half *warpBs = &Bs[warpCol * WN];

        for (int tileRow = 0; tileRow < N_WMMA_ROWS_PER_WARP; tileRow++)
        {
            for (int tileCol = 0; tileCol < N_WMMA_COLS_PER_WARP; tileCol++)
            {
                for (int chunk = 0; chunk < CHUNK_K; chunk++)
                {
                    wmma::load_matrix_sync(aFrag, warpAs + CHUNK_K * WMMA_K, BK);
                    wmma::load_matrix_sync(bFrag, warpBs + CHUNK_K * WMMA_K * BN, BN);

                    wmma::mma_sync(cFrag[tileRow][tileCol], aFrag, bFrag, cFrag[tileRow][tileCol]);
                }
                warpBs += WMMA_N;
            }
            warpAs += WMMA_M * BK;
        }
    }
}

template<int version>
void matmul_Oootomo(float *A, float *B, float *C, int M, int K, int N) 
{
    assert((M % 16) == 0);
    assert((K % 16) == 0);
    assert((N % 16) == 0);

    PROFILE_FUNCTION_SEGMENT_START("allocate gpu");
    
    int AElems = M * K;
    int BElems = K * N;
    int CElems = M * N;
    float *deviceAFull, *deviceBFull, *deviceCFull;
    hipMalloc(&deviceAFull, AElems * sizeof(float));
    hipMalloc(&deviceBFull, BElems * sizeof(float));
    hipMalloc(&deviceCFull, CElems * sizeof(float));

    // these identifiers are just outside the if because otherwise, compilation does not work
    half *deviceA[2], *deviceB[2];
    // {AB, dAB, AdB, dAdB}
    float *deviceC[4];
    if constexpr(version == 1)
    {
        for(int i = 0; i < 2; i++)
        {
            hipMalloc(&deviceA[i], AElems * sizeof(half));
            hipMalloc(&deviceB[i], BElems * sizeof(half));
        }
        for(int i = 0; i < 4; i++)
            hipMalloc(&deviceC[i], CElems * sizeof(float));
    }

    PROFILE_SEGMENTS_SWITCH("memcpy host2device");
    hipMemcpy(deviceAFull, A, AElems * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceBFull, B, BElems * sizeof(float), hipMemcpyHostToDevice);

    if constexpr(version == 1)
    {
        PROFILE_SEGMENTS_SWITCH("split");
        int threadsPerBlock = 256;
        split_cuda<<<M * K / threadsPerBlock, threadsPerBlock>>>(deviceAFull, deviceA[0], deviceA[1]);
        split_cuda<<<K * N / threadsPerBlock, threadsPerBlock>>>(deviceBFull, deviceB[0], deviceB[1]);

        hipDeviceSynchronize();
    }

    PROFILE_SEGMENTS_SWITCH("matmul");
    if constexpr(version == 1)
    {
        // If this is changed, ensure that shared memory is still large enough or 
        // change shared memory settings with CUDA runtime API
        constexpr int CHUNK_K = 2;
        constexpr int BM = WMMA_M * 8;
        constexpr int BN = WMMA_N * 8;
        constexpr int BK = WMMA_K * CHUNK_K;
        // The number of blocks must deivide the matrix dimensions
        assert(M % BM == 0);
        assert(N % BN == 0);
        assert(K % BK == 0);
        constexpr int WM = WMMA_M * 2;
        constexpr int WN = WMMA_N * 2;
        // the number of warps must divide the warp dimensions
        static_assert(BM % WM == 0);
        static_assert(BN % WN == 0);
        constexpr int N_WARP_ROWS_PER_BLOCK = BM / WM;
        constexpr int N_WARP_COLS_PER_BLOCK = BN / WN;
        constexpr int N_WMMA_ROWS_PER_WARP = WM / WMMA_M;
        constexpr int N_WMMA_COLS_PER_WARP = WN / WMMA_N;
        constexpr int threadsPerBlock = N_WARP_ROWS_PER_BLOCK * N_WARP_COLS_PER_BLOCK * WARP_SIZE;
        // In each SMEM loading iteration, each thread loads 4 values from GMEM
        // These asserts ensures that the loading loop does not convert divergent branches (i.e. each thread has 
        // the same amount of values to load)
        static_assert((BM * BK) % (4 * threadsPerBlock) == 0);
        static_assert((BK * BN) % (4 * threadsPerBlock) == 0);
        // These asserts ensure that in each SMEM loading iteration, the threads load N entire rows (and not a half row or something)
        // of the shared memory
        static_assert((4 * threadsPerBlock) % BK == 0);
        static_assert((4 * threadsPerBlock) % BN == 0);
        dim3 blocks(M / BM, N / BN);
        matmul_v1_kernel<BM, BN, BK, WM, WN, CHUNK_K, N_WARP_ROWS_PER_BLOCK, N_WARP_COLS_PER_BLOCK, N_WMMA_ROWS_PER_WARP, N_WMMA_COLS_PER_WARP>
            <<<blocks, threadsPerBlock>>>(deviceA[0], deviceB[0], deviceC[0], M, K, N);
        matmul_v1_kernel<BM, BN, BK, WM, WN, CHUNK_K, N_WARP_ROWS_PER_BLOCK, N_WARP_COLS_PER_BLOCK, N_WMMA_ROWS_PER_WARP, N_WMMA_COLS_PER_WARP>
            <<<blocks, threadsPerBlock>>>(deviceA[1], deviceB[0], deviceC[1], M, K, N);
        matmul_v1_kernel<BM, BN, BK, WM, WN, CHUNK_K, N_WARP_ROWS_PER_BLOCK, N_WARP_COLS_PER_BLOCK, N_WMMA_ROWS_PER_WARP, N_WMMA_COLS_PER_WARP>
            <<<blocks, threadsPerBlock>>>(deviceA[0], deviceB[1], deviceC[2], M, K, N);
        matmul_v1_kernel<BM, BN, BK, WM, WN, CHUNK_K, N_WARP_ROWS_PER_BLOCK, N_WARP_COLS_PER_BLOCK, N_WMMA_ROWS_PER_WARP, N_WMMA_COLS_PER_WARP>
            <<<blocks, threadsPerBlock>>>(deviceA[1], deviceB[1], deviceC[3], M, K, N);
    } 
    else if (version == 2)
    {
        // ...
    }

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    if constexpr(version == 1)
    {
        PROFILE_SEGMENTS_SWITCH("merge");
        int threadsPerBlock = 256;
        merge_cuda<<<M * K / threadsPerBlock, threadsPerBlock>>>(deviceCFull, deviceC[0], deviceC[1], deviceC[2], deviceC[3]);

        hipDeviceSynchronize();
    }

    PROFILE_SEGMENTS_SWITCH("memcpy device2host");
    hipMemcpy(C, deviceCFull, CElems * sizeof(float), hipMemcpyDeviceToHost);


    PROFILE_SEGMENTS_SWITCH("free");
    hipFree(deviceAFull);
    hipFree(deviceBFull);
    hipFree(deviceCFull);

    if constexpr(version == 1)
    {
        for(int i = 0; i < 2; i++)
        {
            hipFree(deviceA[i]);
            hipFree(deviceB[i]);
        }
        for(int i = 0; i < 4; i++)
            hipFree(deviceC[i]);
    }

    PROFILE_SEGMENT_FUNCTION_END();
}

// Note: Does currently not yet work for matrices smaller than 256x256
void matmul_Oootomo_v1(float *A, float *B, float *C, int M, int K, int N)
{
    matmul_Oootomo<1>(A, B, C, M, K, N);
}

void matmul_Oootomo_v2(float *A, float *B, float *C, int M, int K, int N)
{
    matmul_Oootomo<2>(A, B, C, M, K, N);
}
